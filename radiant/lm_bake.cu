#include "hip/hip_runtime.h"

#include <optix.h>
#include <cuda_helpers.h>
#include <vec_math.h>

#include "lm_bake.h"

extern "C" 
{
__constant__ bake_lm_params_t Params;
}


// Orthonormal basis helper
struct Onb
{
    __forceinline__ __device__ Onb(const float3 &normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3 &p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

// A simple hash function
static __inline__ __device__ unsigned int hash(unsigned int a) 
{
    a = (a ^ 61u) ^ (a >> 16u);
    a *= 9u;
    a = a ^ (a >> 4u);
    a *= 0x27d4eb2du;
    a = a ^ (a >> 15u);
    return a;
}

// A simple linear congruential generator for pseudo-random numbers.
static __forceinline__ __device__ float rnd(unsigned int &seed) 
{
    seed = 1664525u * seed + 1013904223u;
    return float(seed & 0x00FFFFFF) / float(0x01000000);
}

// Cosine-weighted hemisphere sampling in local space.
static __forceinline__ __device__ float3 cosine_sample_hemisphere(unsigned int &seed) 
{
    float u1 = rnd(seed);
    float u2 = rnd(seed);
    float r = sqrtf(u1);
    float theta = 2.0f * M_PIf * u2;
    float x = r * cosf(theta);
    float y = r * sinf(theta);
    float z = sqrtf(1.0f - u1);
    return make_float3(x, y, z);
}

static __device__ float CalculateDirectionalLight(float3 Position, float3 SurfaceNormal)
{
    float LightValue = 0.f;

    if (Params.DoDirectionalLight)
    {
        float CosTheta = dot(SurfaceNormal, Params.DirectionToSun);
        if (CosTheta > 0.f)
        {
            // Send
            // p0: SurfaceNormal.x
            // p1: SurfaceNormal.y
            // p2: SurfaceNormal.z
            // Receive
            // p0: float light contribution
            unsigned int p0, p1, p2;
            p0 = __float_as_uint(SurfaceNormal.x);
            p1 = __float_as_uint(SurfaceNormal.y);
            p2 = __float_as_uint(SurfaceNormal.z);
            optixTrace( // Trace the ray against our scene hierarchy
                Params.GASHandle,
                Position,
                Params.DirectionToSun,
                0.01f,    // Min intersection distance
                100000.f, // Max intersection distance
                0.0f,     // rayTime -- used for motion blur
                OptixVisibilityMask(255), // Specify always visible
                OPTIX_RAY_FLAG_NONE,
                RAY_TYPE_DIRECTIONAL_LIGHT, // SBT offset   -- See SBT discussion
                RAY_TYPE_COUNT,             // SBT stride   -- See SBT discussion
                RAY_TYPE_DIRECTIONAL_LIGHT, // missSBTIndex -- See SBT discussion
                p0, p1, p2);
            float result = __uint_as_float(p0);
            LightValue += result;
        }
    }

    int CountOfPointLights = Params.CountOfPointLights;
    for (unsigned int i = 0; i < CountOfPointLights; ++i)
    {
        float3 PointLightWorldPos = Params.PointLights[i].Position;
        float3 DirectionToPointLight = normalize(PointLightWorldPos - Position);
        float CosTheta = dot(SurfaceNormal, DirectionToPointLight);
        if (CosTheta > 0.f)
        {
            // Send
            // p0: unsigned int point light index
            // p1: SurfaceNormal.x
            // p2: SurfaceNormal.y
            // p3: SurfaceNormal.z
            // Receive
            // p0: float light contribution
            unsigned int p0, p1, p2, p3;
            p0 = i;
            p1 = __float_as_uint(SurfaceNormal.x);
            p2 = __float_as_uint(SurfaceNormal.y);
            p3 = __float_as_uint(SurfaceNormal.z);
            optixTrace(
                Params.GASHandle,
                Position,
                DirectionToPointLight,
                0.01f,    // Min intersection distance
                100000.f, // Max intersection distance
                0.0f,     // rayTime -- used for motion blur
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,
                RAY_TYPE_POINT_LIGHT,
                RAY_TYPE_COUNT,
                RAY_TYPE_POINT_LIGHT,
                p0, p1, p2, p3);
            float LightContribution = __uint_as_float(p0);
            LightValue += LightContribution;
        }
    }

    return LightValue;
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();

    float3 TexelPosition = Params.TexelWorldPositions[idx.x];
    float3 TexelNormal = Params.TexelWorldNormals[idx.x];

    float DirectLightValue = CalculateDirectionalLight(TexelPosition, TexelNormal);

    // Instead of recursively casting rays, maybe it would be faster if
    // all the texels did monte carlo sampling once, update the light buffer
    // then do the subsequent bounces? by doing monte carlo sampling again.
    // we would need a way to look up or sample the lightmap upon a hit.
    // well, the lightmap is a flat array, so I can find the texel index
    // from the UV by doing V * ActualHeight * ActualWidth + U * ActualWidth
    // or maybe, 

    // Indirect light values
    // Do Monte Carlo sampling from this texel to gather bounce lighting
    unsigned int seed = hash(idx.x); // Each lightmap texel needs a unique seed
    float3 AccumulatedRadiance = make_float3(0.f);
    const int NumSamples = Params.NumberOfSampleRaysPerTexel;
    const int NumBounces = Params.NumberOfBounces;
    // in scenes with tiny slivers of surfaces with direct lighting, doing only one bounce
    // is very high variance and introduces lots of noise. On the other hand, my indoor point
    // light test scene looks good even with low num samples and single bounce.
    int i = NumSamples;
    do
    {
        // If results are a bit noisy, I can try introducing subpixel jitter

        // for loop for bounces
        // need to send: 
        //      r0: seed
        //      r1-3: attenuation thus far
        // need to receive: 
        //      r0: updated seed
        //      r1-3: hit point position x y z
        //      r4-6: hit point normal
        //      r7: a "done" flag if we miss
        //      r8-10: the radiance from that direction (which is direct light at that point * attenuation)
        //      r11-13: the updated attenuation
        // upon return, if either attenuation is too weak or we missed, we stop.
        // should we put an upper limit on bounces too? or let it run until atten is weak?

        float3 Attenuation = make_float3(1.f);
        float3 SampleRayOrigin = TexelPosition;
        float3 NormalAtRayOrigin = TexelNormal;
        int DoneFlag = false;
        for (int j = 0; j < NumBounces; ++j)
        {
            float3 SampleRayDirection = cosine_sample_hemisphere(seed);
            Onb onb(NormalAtRayOrigin);
            onb.inverse_transform(SampleRayDirection);
            // at this point, SampleRayDirection is in world space

            unsigned int r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13;
            // r0 = seed;
            r1 = __float_as_uint(Attenuation.x);
            r2 = __float_as_uint(Attenuation.y);
            r3 = __float_as_uint(Attenuation.z);
            optixTrace(
                Params.GASHandle,
                SampleRayOrigin,
                SampleRayDirection,
                0.01f,
                100000.f,
                0.0f,
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,
                RAY_TYPE_HEMISPHERE_SAMPLE,
                RAY_TYPE_COUNT,
                RAY_TYPE_HEMISPHERE_SAMPLE,
                r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13);
            // seed = r0;
            SampleRayOrigin.x = __uint_as_float(r1);
            SampleRayOrigin.y = __uint_as_float(r2);
            SampleRayOrigin.z = __uint_as_float(r3);
            NormalAtRayOrigin.x = __uint_as_float(r4);
            NormalAtRayOrigin.y = __uint_as_float(r5);
            NormalAtRayOrigin.z = __uint_as_float(r6);
            DoneFlag = r7;
            float3 Radiance;
            Radiance.x = __uint_as_float(r8);
            Radiance.y = __uint_as_float(r9);
            Radiance.z = __uint_as_float(r10);
            Attenuation.x = __uint_as_float(r11);
            Attenuation.y = __uint_as_float(r12);
            Attenuation.z = __uint_as_float(r13);

            if (DoneFlag)
                break;

            float3 AttenuatedRadiance = Radiance * Attenuation;
            AccumulatedRadiance += AttenuatedRadiance;
        }

    } while (--i);
    // Only using one component lighting for now
    float IrradianceAtThisPoint = AccumulatedRadiance.x / float(NumSamples);

    Params.OutputLightmap[idx.x] = DirectLightValue + IrradianceAtThisPoint;
    // Params.OutputLightmap[idx.x] = IrradianceAtThisPoint;

}

extern "C" __global__ void __miss__HemisphereSample()
{
    optixSetPayload_7(true);
}

extern "C" __global__ void __closesthit__HemisphereSample()
{
    float3 Attenuation;
    Attenuation.x = __uint_as_float(optixGetPayload_1());
    Attenuation.y = __uint_as_float(optixGetPayload_2());
    Attenuation.z = __uint_as_float(optixGetPayload_3());

    float3 RayOrigin = optixGetWorldRayOrigin();
    float3 RayDirection = optixGetWorldRayDirection();
    float HitT = optixGetRayTmax();
    float3 HitPointPosition = RayOrigin + HitT * RayDirection; // Compute the world-space hit position

    float3 Vertices[3] = {}; // vertices of the hit triangle
    optixGetTriangleVertexData(Params.GASHandle, optixGetPrimitiveIndex(), optixGetSbtGASIndex(), 0.f, Vertices);
    float3 HitPointNormal = normalize(cross(Vertices[1] - Vertices[0], Vertices[2] - Vertices[0]));

    if (dot(HitPointNormal, RayDirection) >= 0.f)
    {
        // if the hemisphere sampling ray hits a backface, then this point is inside a wall
        optixSetPayload_7(true);
    }
    else
    {
        float DirectLightValueAtHitPoint = CalculateDirectionalLight(HitPointPosition, HitPointNormal);
        // TODO the Optix path tracer sample uses weighted luminance which is specifically 30% red, 59% green, 11% blue
        float3 Albedo = make_float3(0.34f);
        Attenuation *= Albedo;

        optixSetPayload_1(__float_as_uint(HitPointPosition.x));
        optixSetPayload_2(__float_as_uint(HitPointPosition.y));
        optixSetPayload_3(__float_as_uint(HitPointPosition.z));
        optixSetPayload_4(__float_as_uint(HitPointNormal.x));
        optixSetPayload_5(__float_as_uint(HitPointNormal.y));
        optixSetPayload_6(__float_as_uint(HitPointNormal.z));
        optixSetPayload_7(__float_as_uint(false));
        optixSetPayload_8(__float_as_uint(DirectLightValueAtHitPoint));
        optixSetPayload_9(__float_as_uint(DirectLightValueAtHitPoint));
        optixSetPayload_10(__float_as_uint(DirectLightValueAtHitPoint));
        optixSetPayload_11(__float_as_uint(Albedo.x));
        optixSetPayload_12(__float_as_uint(Albedo.y));
        optixSetPayload_13(__float_as_uint(Albedo.z));
    }
}

extern "C" __global__ void __miss__PointLight()
{
    // If ray misses and goes off into space, then nothing in between the texel and the point light

    float3 SurfaceNormal;
    SurfaceNormal.x = __uint_as_float(optixGetPayload_1());
    SurfaceNormal.y = __uint_as_float(optixGetPayload_2());
    SurfaceNormal.z = __uint_as_float(optixGetPayload_3());

    unsigned int PointLightIdx = optixGetPayload_0();
    cu_pointlight_t PointLight = Params.PointLights[PointLightIdx];
    float3 ToLight = PointLight.Position - optixGetWorldRayOrigin();
    float DistToLight = length(ToLight);

    float CosTheta = dot(SurfaceNormal, normalize(ToLight));
    float AttenLin = PointLight.AttenuationLinear;
    float AttenQuad = PointLight.AttenuationQuadratic;
    float Attenuation = 1.f / 
        (1.f + AttenLin * DistToLight + AttenQuad * DistToLight * DistToLight);
    float DirectIntensity = CosTheta * Attenuation;

    optixSetPayload_0(__float_as_uint(DirectIntensity));
}

extern "C" __global__ void __closesthit__PointLight()
{
    // If ray hits an object, then check if point light is before or after this object

    float3 SurfaceNormal;
    SurfaceNormal.x = __uint_as_float(optixGetPayload_1());
    SurfaceNormal.y = __uint_as_float(optixGetPayload_2());
    SurfaceNormal.z = __uint_as_float(optixGetPayload_3());

    unsigned int PointLightIdx = optixGetPayload_0();
    cu_pointlight_t PointLight = Params.PointLights[PointLightIdx];
    float3 PointLightWorldPos = PointLight.Position;

    float3 RayOrigin = optixGetWorldRayOrigin();
    float3 RayDirection = optixGetWorldRayDirection();
    float HitT = optixGetRayTmax();
    float3 HitPosition = RayOrigin + HitT * RayDirection; // Compute the world-space hit position

    float3 ToLight = PointLightWorldPos - RayOrigin;
    float DistToLight = length(ToLight);
    float DistToHit = length(HitPosition - RayOrigin);

    if (DistToLight < DistToHit)
    {
        float CosTheta = dot(SurfaceNormal, normalize(ToLight));
        float AttenLin = PointLight.AttenuationLinear;
        float AttenQuad = PointLight.AttenuationQuadratic;
        float Attenuation = 1.f / 
            (1.f + AttenLin * DistToLight + AttenQuad * DistToLight * DistToLight);
        float DirectIntensity = CosTheta * Attenuation;

        optixSetPayload_0(__float_as_uint(DirectIntensity));
    }
    else
    {
        optixSetPayload_0(__float_as_uint(0.f));
    }
}

extern "C" __global__ void __miss__DirectionalLight()
{
    float3 SurfaceNormal;
    SurfaceNormal.x = __uint_as_float(optixGetPayload_0());
    SurfaceNormal.y = __uint_as_float(optixGetPayload_1());
    SurfaceNormal.z = __uint_as_float(optixGetPayload_2());

    float DirectIntensity = dot(SurfaceNormal, Params.DirectionToSun);
    optixSetPayload_0(__float_as_uint(DirectIntensity));
}

extern "C" __global__ void __closesthit__DirectionalLight()
{
    optixSetPayload_0(__float_as_uint(0.f));
}
